#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <hip/hip_runtime.h>
#include "cuda.cuh"

// CUDA error checking macro
#define CUDA_CHECK(call) \
  do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
      exit(EXIT_FAILURE); \
    } \
  } while(0)

__host__ float* cpu_to_cuda(float* cpu_data, int device_id, size_t size) {
  if (cpu_data == NULL) {
    fprintf(stderr, "CPU data pointer is null!\n");
    exit(EXIT_FAILURE);
  }

  // setting the specified GPU device
  CUDA_CHECK(hipSetDevice(device_id));

  // allocating GPU memory for float32 data
  float* gpu_float_data;
  size_t data_size = size * sizeof(float);
  CUDA_CHECK(hipMalloc((void**)&gpu_float_data, data_size));

  // copying float32 data from CPU to GPU
  CUDA_CHECK(hipMemcpy(gpu_float_data, cpu_data, data_size, hipMemcpyHostToDevice));
  free(cpu_data);  // cleaning up temporary CPU float data
  // printf("float data moved to GPU device %d as float32 array\n", device_id);
  return gpu_float_data;
}

__host__ float* cuda_to_cpu(float* gpu_data, size_t size) {
  if (gpu_data == NULL || !size) {
    fprintf(stderr, "Invalid input parameters!\n");
    exit(EXIT_FAILURE);
  }

  // checking if data is actually on GPU
  hipPointerAttribute_t attributes;
  hipError_t error = hipPointerGetAttributes(&attributes, gpu_data);
  if (error != hipSuccess || attributes.type != hipMemoryTypeDevice) {
    fprintf(stderr, "Data is not on GPU or invalid pointer!\n");
    exit(EXIT_FAILURE);
  }

  // allocating CPU memory for float32 data
  float* cpu_data = (float*)malloc(size * sizeof(float));
  if (cpu_data == NULL) {
    fprintf(stderr, "Memory allocation failed for CPU float data!\n");
    exit(EXIT_FAILURE);
  }

  // copying float32 data from GPU to CPU
  CUDA_CHECK(hipMemcpy(cpu_data, gpu_data, size * sizeof(float), hipMemcpyDeviceToHost));
  // printf("GPU float32 data converted to CPU\n");
  return cpu_data;
}

__host__ void free_cuda(float* data) {
  if (data == NULL) {
    fprintf(stderr, "Cannot free null GPU data pointer!\n");
    return;
  }

  // checking if the pointer is actually on GPU
  hipPointerAttribute_t attributes;
  hipError_t error = hipPointerGetAttributes(&attributes, data);
  if (error != hipSuccess) {
    fprintf(stderr, "Invalid GPU pointer or error checking pointer attributes!\n");
    return;
  }

  if (attributes.type != hipMemoryTypeDevice) {
    fprintf(stderr, "Pointer is not GPU memory!\n");
    return;
  }

  // freeing the GPU memory
  CUDA_CHECK(hipFree(data));
  printf("GPU memory freed\n");
}

// utility function to get GPU device count
__host__ int get_cuda_device_count(void) {
  int device_count = 0;
  hipError_t error = hipGetDeviceCount(&device_count);
  if (error != hipSuccess) {
    fprintf(stderr, "Failed to get CUDA device count: %s\n", hipGetErrorString(error));
    return 0;
  }
  return device_count;
}

// utility function to print GPU device information
__host__ void print_cuda_device_info(int device_id) {
  hipDeviceProp_t prop;
  CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
  
  printf("GPU Device %d: %s\n", device_id, prop.name);
  printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
  printf("  Total Global Memory: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
  printf("  Multiprocessors: %d\n", prop.multiProcessorCount);
  printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
  printf("  Max Threads per Multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
  printf("  Warp Size: %d\n", prop.warpSize);
}

__device__ size_t calculate_flat_index_cuda(int* indices, int* strides, size_t ndim) {
  size_t flat_idx = 0;
  for (size_t i = 0; i < ndim; i++) {
    flat_idx += indices[i] * strides[i];
  }
  return flat_idx;
}

__device__ void flat_to_multi_index_cuda(size_t flat_idx, int* shape, size_t ndim, int* indices) {
  for (int i = ndim - 1; i >= 0; i--) {
    indices[i] = flat_idx % shape[i];
    flat_idx /= shape[i];
  }
}

__global__ void contiguous_tensor_cuda(void* src_data, void* dst_data, int* strides, int* shape, size_t ndim, size_t elem_size, size_t total_size) {
  size_t flat_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (flat_idx >= total_size) return;

  extern __shared__ int shared[];
  int* indices = shared + threadIdx.x * ndim;

  flat_to_multi_index_cuda(flat_idx, shape, ndim, indices);
  size_t src_offset = 0;
  for (size_t dim = 0; dim < ndim; dim++) {
    src_offset += indices[dim] * strides[dim] * elem_size;
  }

  char* src = (char*)src_data;
  char* dst = (char*)dst_data;
  size_t dst_offset = flat_idx * elem_size;
  for (size_t i = 0; i < elem_size; i++) {
    dst[dst_offset + i] = src[src_offset + i];
  }
}

__global__ void __assign_tensor_kernel__(float* a, float* out, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    out[i] = a[i];
  }
}

__host__ void assign_tensor_cuda(float* a, float* out, size_t size) {
  int n_of_blocks = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  __assign_tensor_kernel__<<<n_of_blocks, THREADS_PER_BLOCK>>>(a, out, size);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
  hipDeviceSynchronize();
}